#include "hip/hip_runtime.h"
#include "renderer.cuh"

Circle* generateCircles(std::size_t n, int width, int height, int minRadius, int maxRadius) {
    auto* circles = new Circle[n];
    std::mt19937 generator(777);

    std::uniform_int_distribution<int> colorDistribution(0, 255);
    std::uniform_int_distribution<int> pointXDistribution(1, width);
    std::uniform_int_distribution<int> pointYDistribution(1, height);
    std::uniform_int_distribution<int> radiusDistribution(minRadius, maxRadius);

#pragma omp parallel for default(none) shared(circles, generator)
    for (int i = 0; i < n; i++) {
        cv::Scalar color(colorDistribution(generator), colorDistribution(generator), colorDistribution(generator), 255);
        cv::Point center(pointXDistribution(generator), pointYDistribution(generator));
        int r = radiusDistribution(generator);
        circles[i] = Circle{color, center, r};
    }

    return circles;
}

double sequentialRenderer(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

    for (int i = 0; i < nPlanes; i++) {
        planes[i] = TRANSPARENT_MAT;
        for (int j = 0; j < nCircles; j++) {
            auto circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = sequentialCombinePlanes(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END

    delete[] planes;

    cv::imwrite(SEQ_IMG_PATH + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat sequentialCombinePlanes(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result = TRANSPARENT_MAT;
    int cn = result.channels();
    for (int i = 0; i < result.rows; i++) {
        for (int j = 0; j < result.cols; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double parallelRenderer(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = TRANSPARENT_MAT;
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = parallelCombinePlanes(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END

    delete[] planes;

    cv::imwrite(PAR_IMG_PATH + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat parallelCombinePlanes(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result = TRANSPARENT_MAT;
    int cn = result.channels();
#pragma omp parallel for default(none) shared(result, planes) firstprivate(nPlanes, cn) collapse(2)
    for (int i = 0; i < result.rows; i++) {
        for (int j = 0; j < result.cols; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double cudaRenderer(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = TRANSPARENT_MAT;
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = cudaCombinePlanes(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END

    delete[] planes;

    cv::imwrite(CUDA_IMG_PATH + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat cudaCombinePlanes(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result = TRANSPARENT_MAT;
    int width = result.cols;
    int height = result.rows;

    uchar4* d_resultData;
    uchar4* d_planesData;

    // INITIALIZATION OF GPU MEMORY
    hipMalloc((void**)&d_resultData, width * height * sizeof(uchar4));
    hipMalloc((void**)&d_planesData, width * height * sizeof(uchar4) * nPlanes);

    hipMemcpy(d_resultData, result.data, width * height * sizeof(uchar4), hipMemcpyHostToDevice);
    for (std::size_t i = 0; i < nPlanes; i++)
        hipMemcpy(d_planesData + i * width * height, planes[i].data, width * height * sizeof(uchar4), hipMemcpyHostToDevice);

    // GRID AND BLOCK DIMENSIONS
    dim3 block(16, 16);
    dim3 grid((result.cols + block.x - 1) / block.x, (result.rows + block.y - 1) / block.y);

    // CUDA KERNEL
    cudaKernelCombinePlanes<<<grid, block>>>(d_resultData, d_planesData, result.cols, result.rows, (int) nPlanes);
    hipDeviceSynchronize();

    // COPY RESULT FROM GPU TO CPU
    hipMemcpy(result.data, d_resultData, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);

    // FREE MEMORY
    hipFree(d_planesData);
    hipFree(d_resultData);

    return result;
}

__global__ void cudaKernelCombinePlanes(uchar4* resultData, const uchar4* planesData, int width, int height, int nPlanes) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        auto idx = y * width + x;
        for (int z = 0; z < nPlanes; z++) {
            auto idxP = z * width * height + idx;
            resultData[idx].x = resultData[idx].x * (1.0f - ALPHA) + planesData[idxP].x * ALPHA;
            resultData[idx].y = resultData[idx].y * (1.0f - ALPHA) + planesData[idxP].y * ALPHA;
            resultData[idx].z = resultData[idx].z * (1.0f - ALPHA) + planesData[idxP].z * ALPHA;
            resultData[idx].w = resultData[idx].w * (1.0f - ALPHA) + planesData[idxP].w * ALPHA;
        }
    }
}

