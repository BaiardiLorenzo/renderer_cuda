#include "src/renderer.cuh"
#include "src/test.h"
#include "src/utils.h"
#include <map>
#include <iomanip>

void testParallelization(const std::vector<std::size_t>& testPlanes){
    headerResults(RESULT_PATH, omp_get_num_procs());
    for (auto test: testPlanes) {
        // GENERATION OF CIRCLES
        auto circles = parallelGenerateCircles(test * N_CIRCLES, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
        auto planes = parallelGeneratePlanes(test, circles, N_CIRCLES);

        printf("\nTEST PLANES: %llu\n", test);

        // TEST SEQUENTIAL
        double tSeq = sequentialRenderer(planes, test);
        printf("SEQUENTIAL Time: %f\n", tSeq);

        // TEST OPENMP
        std::map<std::size_t, double> tPars;
        std::map<std::size_t, double> speedUps;
        for (int i=2; i<=omp_get_num_procs(); i+=2) {
            // SET NUMBER OF THREADS
            omp_set_num_threads(i);

            // TEST PARALLEL
            double tPar = parallelRenderer(planes, test);
            printf("PARALLEL-%d Time: %f\n", i, tPar);

            double speedUp = tSeq / tPar;
            printf("PARALLEL-%d Speedup: %f \n", i, speedUp);

            // SAVE RESULTS
            tPars.insert(std::pair<std::size_t, double>(i, tPar));
            speedUps.insert(std::pair<std::size_t, double>(i, speedUp));
        }

        // TEST CUDA
        double tCuda = cudaRenderer(planes, test);
        printf("CUDA Time: %f\n", tCuda);

        double speedUpCuda = tSeq / tCuda;
        printf("CUDA Speedup: %f\n", speedUpCuda);

        // TEST CUDA COLOR
        double tCudaColor = cudaRendererColor(planes, test);
        printf("CUDA-COLOR Time: %f\n", tCudaColor);

        double speedUpCudaColor = tSeq / tCudaColor;
        printf("CUDA-COLOR Speedup: %f\n\n", speedUpCudaColor);

        // WRITE RESULTS TO TXT FILE
        exportResults(RESULT_PATH, test, tSeq, tPars, speedUps, tCuda, speedUpCuda, tCudaColor, speedUpCudaColor);

        // DELETE ARRAY DYNAMIC ALLOCATED
        delete[] circles;
        delete[] planes;
    }
}

void testCudaMemcpy(const std::vector<std::size_t>& testPlanes){
    headerResultsMemcpy(RESULT_MEMCPY_PATH);
    printf("\nTEST CUDA MEMCPY\n");
    for (auto test: testPlanes) {
        printf("TEST PLANES: %llu\n", test);

        // GENERATION OF CIRCLES
        auto circles = parallelGenerateCircles(test * N_CIRCLES, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
        auto planes = parallelGeneratePlanes(test, circles, N_CIRCLES);

        // TEST SEQUENTIAL
        double tSeq = sequentialRenderer(planes, test);
        printf("SEQUENTIAL Time: %f\n", tSeq);

        // TEST CUDA BLOCKS
        double tCuda = cudaRenderer(planes, test);
        printf("CUDA Time: %f\n", tCuda);

        double speedUpCuda = tSeq / tCuda;
        printf("CUDA Speedup: %f\n\n", speedUpCuda);

        double tCudaMemcpy = cudaRendererCopy(planes, test);
        printf("CUDA-MEMCPY Time: %f\n", tCudaMemcpy);

        double speedUpCudaMemcpy = tSeq / tCudaMemcpy;
        printf("CUDA-MEMCPY Speedup: %f\n\n", speedUpCudaMemcpy);

        // WRITE RESULTS TO TXT FILE
        exportResultsMemcpy(RESULT_MEMCPY_PATH, test, tSeq, tCuda, speedUpCuda, tCudaMemcpy, speedUpCudaMemcpy);

        // DELETE ARRAY DYNAMIC ALLOCATED
        delete[] circles;
        delete[] planes;
    }
}

void testCudaBlocks(const std::vector<std::size_t>& testPlanes){
    headerResultsBlocks(RESULT_BLOCKS_PATH);
    printf("\nTEST CUDA GRID\n");
    for (auto test: testPlanes) {
        printf("TEST PLANES: %llu\n", test);

        // GENERATION OF CIRCLES
        auto circles = parallelGenerateCircles(test * N_CIRCLES, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
        auto planes = parallelGeneratePlanes(test, circles, N_CIRCLES);

        // TEST SEQUENTIAL
        double tSeq = sequentialRenderer(planes, test);
        printf("SEQUENTIAL Time: %f\n", tSeq);

        // TEST CUDA BLOCKS
        double tBlock8 = cudaRenderer(planes, test, 8);
        printf("CUDA 8x8 Time: %f\n", tBlock8);

        double tBlock16 = cudaRenderer(planes, test, 16);
        printf("CUDA 16x16 Time: %f\n", tBlock16);

        double tBlock32 = cudaRenderer(planes, test, 32);
        printf("CUDA 32x32 Time: %f\n", tBlock32);

        // WRITE RESULTS TO TXT FILE
        exportResultsBlocks(RESULT_BLOCKS_PATH, test, tSeq, tBlock8, tBlock16, tBlock32);

        // DELETE ARRAY DYNAMIC ALLOCATED
        delete[] circles;
        delete[] planes;
    }
}

void testCircles(const std::vector<std::size_t>& testPlanes, const std::vector<std::size_t>& testCircles){
    headerResultsCircle(RESULT_CIRCLES_PATH);
    printf("\nTEST CIRCLES\n");
    for (auto testP: testPlanes) {
        printf("TEST PLANES: %llu\n", testP);

        for (auto testC : testCircles) {
            // GENERATION OF CIRCLES
            double start = omp_get_wtime();
            auto circles = sequentialGenerateCircles(testP * testC, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
            auto planes = sequentialGeneratePlanes(testP, circles, testC);
            double seqTime = omp_get_wtime() - start;
            printf("Sequential Time: %f\n", seqTime);

            // DELETE ARRAY DYNAMIC ALLOCATED
            delete[] circles;
            delete[] planes;

            // GENERATION OF CIRCLES
            start = omp_get_wtime();
            circles = parallelGenerateCircles(testP * testC, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
            planes = parallelGeneratePlanes(testP, circles, testC);
            double parTime = omp_get_wtime() - start;
            printf("Parallel Time: %f\n", parTime);

            // DELETE ARRAY DYNAMIC ALLOCATED
            delete[] circles;
            delete[] planes;

            // WRITE RESULTS TO CSV FILE
            exportResultsCircle(RESULT_CIRCLES_PATH, testP, testC, seqTime, parTime, seqTime/parTime);
        }

    }
}

int main() {
#ifdef _OPENMP
    printf("**OPENMP :: Number of cores/threads: %d**\n", omp_get_num_procs());
    omp_set_dynamic(0);
#endif
    hipDeviceProp_t device{};
    hipGetDeviceProperties(&device, 0);
    printf("**CUDA :: MultiProcessorCount: %d**\n", device.multiProcessorCount); // 6
    printf("**CUDA :: Max Threads per MultiProcessor: %d**\n", device.maxThreadsPerMultiProcessor); // 2048
    printf("**CUDA :: Max Threads per block: %d**\n", device.maxThreadsPerBlock); // 1024
    printf("**CUDA :: Max Blocks per MultiProcessor: %d**\n", device.maxBlocksPerMultiProcessor); // 32
    // MAXTHREADS TOTAL: 12288

    std::vector<std::size_t> testPlanes;
    for (std::size_t i = MIN_TEST; i <= MAX_TESTS; i += SPACE)
        testPlanes.push_back(i);

    testParallelization(testPlanes);

    // N = 1000 - 10000
    // D = 256x256, 512x512, 1024x1024
    //testCudaMemcpy(testPlanes);

    // N = 500, 5000
    // D = 256x256, 512x512, 1024x1024
    //std::vector<std::size_t> testPlanesBlocks {500, 5000};
    //testCudaBlocks(testPlanesBlocks);

    // n = 50, 500
    // N = 100, 1000, 10000
    // D = 256x256, 512x512, 1024x1024
    //std::vector<std::size_t> testPlanesCircles {100, 1000, 10000};
    //std::vector<std::size_t> testCirclesCircles {50, 500};
    //testCircles(testPlanesCircles, testCirclesCircles);

    return 0;
}
