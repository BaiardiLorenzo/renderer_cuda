#include "src/renderer.cuh"
#include "src/test.h"
#include <map>
#include <iomanip>

void headerResults(const std::string& filename, int nThreads){
    std::ofstream outfile;
    outfile.open(filename);
    if(outfile.is_open())
        outfile << "TEST;T_SEQ;";
    for(int i=2; i<=nThreads; i+=2)
        outfile << "T_PAR" << i << ";SPEEDUP" << i << ";";
    outfile << "T_CUDA;SPEEDUP_CUDA;T_CUDA_COLOR;SPEEDUP_CUDA_COLOR\n";
    outfile.close();
}

void exportResults(const std::string& filename, std::size_t test, double tSeq, const std::map<std::size_t, double>& tPars,
                   std::map<std::size_t,double> speedUps, double tCuda, double speedUpCuda, double tCudaColor=-1, double speedUpCudaColor=-1){
    std::ofstream outfile;
    outfile.open(filename, std::ios::out | std::ios::app);
    if(outfile.is_open()){
        outfile << std::fixed << std::setprecision(3);
        outfile << test << ";" << tSeq << ";";
        for(auto tPar: tPars)
            outfile << tPar.second << ";" << speedUps[tPar.first] << ";";
        outfile << tCuda << ";" << speedUpCuda << ";" << tCudaColor << ";" << speedUpCudaColor << "\n";
    }
    outfile.close();
}


int main() {
#ifdef _OPENMP
    printf("**OPENMP :: Number of cores/threads: %d**\n", omp_get_num_procs());
    omp_set_dynamic(0);
#endif
    hipDeviceProp_t device{};
    hipGetDeviceProperties(&device, 0);
    printf("**CUDA :: MultiProcessorCount: %d**\n", device.multiProcessorCount); // 6
    printf("**CUDA :: Max Threads per MultiProcessor: %d**\n", device.maxThreadsPerMultiProcessor); // 2048
    printf("**CUDA :: Max Threads per block: %d**\n", device.maxThreadsPerBlock); // 1024
    printf("**CUDA :: Max Blocks per MultiProcessor: %d**\n", device.maxBlocksPerMultiProcessor); // 32
    // MAXTHREADS TOTAL: 12288


    headerResults(RESULT_PATH, omp_get_num_procs());
    std::vector<std::size_t> testPlanes;
    for (std::size_t i = MIN_TEST; i <= MAX_TESTS; i += SPACE)
        testPlanes.push_back(i);

    for (auto test: testPlanes) {
        // GENERATION OF CIRCLES
        auto circles = generateCircles(test * N_CIRCLES, WIDTH, HEIGHT, MIN_RADIUS, MAX_RADIUS);
        auto planes = generatePlanes(test, circles, N_CIRCLES);

        printf("\nTEST PLANES: %llu\n", test);

        // TEST SEQUENTIAL
        double tSeq = sequentialRenderer(planes, test);
        printf("SEQUENTIAL Time: %f\n", tSeq);
/*
        // TEST PARALLEL
        std::map<std::size_t, double> tPars;
        std::map<std::size_t, double> speedUps;
        for (int i=2; i<=omp_get_num_procs(); i+=2) {
            // SET NUMBER OF THREADS
            omp_set_num_threads(i);

            // TEST PARALLEL
            double tPar = parallelRenderer(planes, test);
            printf("PARALLEL-%d Time: %f\n", i, tPar);

            double speedUp = tSeq / tPar;
            printf("PARALLEL-%d Speedup: %f \n", i, speedUp);

            // SAVE RESULTS
            tPars.insert(std::pair<std::size_t, double>(i, tPar));
            speedUps.insert(std::pair<std::size_t, double>(i, speedUp));
        }
        */

        // TEST CUDA
        double tCuda = cudaRenderer(planes, test);
        printf("CUDA Time: %f\n", tCuda);

        double speedUpCuda = tSeq / tCuda;
        printf("CUDA Speedup: %f\n", speedUpCuda);

        // TEST CUDA COLOR
        double tCudaColor = cudaRendererColor(planes, test);
        printf("CUDA-COLOR Time: %f\n", tCudaColor);

        double speedUpCudaColor = tSeq / tCudaColor;
        printf("CUDA-COLOR Speedup: %f\n\n", speedUpCudaColor);

        // WRITE RESULTS TO TXT FILE
        //exportResults(RESULT_PATH, test, tSeq, tPars, speedUps, tCuda, speedUpCuda, tCudaColor, speedUpCudaColor);

        // DELETE ARRAY DYNAMIC ALLOCATED
        delete[] circles;
        delete[] planes;
    }
    return 0;
}
