#include "hip/hip_runtime.h"
#include "renderer.cuh"

Circle* generateCircles(std::size_t n) {
    auto* circles = new Circle[n];
    std::mt19937 generator(777);

    std::uniform_int_distribution<int> colorDistribution(0, 255);
    std::uniform_int_distribution<int> pointXDistribution(1, WIDTH);
    std::uniform_int_distribution<int> pointYDistribution(1, HEIGHT);
    std::uniform_int_distribution<int> radiusDistribution(MIN_RADIUS, MAX_RADIUS);

#pragma omp parallel for default(none) shared(circles, generator)
    for (int i = 0; i < n; i++) {
        cv::Scalar color(colorDistribution(generator), colorDistribution(generator), colorDistribution(generator), 255);
        cv::Point center(pointXDistribution(generator), pointYDistribution(generator));
        int r = radiusDistribution(generator);
        circles[i] = Circle{color, center, r};
    }

    return circles;
}

double rendererSequential(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            auto circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesSequential(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END

    printf("Sequential time %f sec.\n", time);

    delete[] planes;

    cv::imwrite("../img/seq_" + std::to_string(nPlanes) + ".png", result);
    // cv::waitKey(0);
    return time;
}

cv::Mat combinePlanesSequential(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
    int cn = result.channels();
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double rendererParallel(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesParallel(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END
    printf("Parallel time %f sec.\n", time);

    delete[] planes;

    cv::imwrite("../img/par_" + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat combinePlanesParallel(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
    int cn = result.channels();
#pragma omp parallel for default(none) shared(result, planes) firstprivate(nPlanes, cn) collapse(2)
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double rendererCuda(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesCuda(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END
    printf("Cuda time %f sec.\n", time);

    delete[] planes;

    // cv::imshow("TEST", result);
    // cv::waitKey(0);

    cv::imwrite("../img/cuda_" + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat combinePlanesCuda(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);

    uchar4* d_resultData;
    uchar4* d_planesData;

    // Initialize pointers on GPU
    hipMalloc((void**)&d_resultData, WIDTH * HEIGHT * sizeof(uchar4));
    hipMalloc((void**)&d_planesData, WIDTH * HEIGHT * sizeof(uchar4) * nPlanes);

    hipMemcpy(d_resultData, result.data, WIDTH * HEIGHT * sizeof(uchar4), hipMemcpyHostToDevice);
    for (std::size_t i = 0; i < nPlanes; i++) {
        uchar4* d_plane = d_planesData + i * WIDTH * HEIGHT;
        hipMemcpy(d_plane, planes[i].data, WIDTH * HEIGHT * sizeof(uchar4), hipMemcpyHostToDevice);
    }

    // GRID AND BLOCK DIMENSIONS
    dim3 block(16, 16);
    dim3 grid((result.cols + block.x - 1) / block.x, (result.rows + block.y - 1) / block.y);

    // CUDA KERNEL
    combinePlanesKernel<<<grid, block>>>(d_resultData, d_planesData, result.cols, result.rows, (int)nPlanes);
    hipDeviceSynchronize();

    // COPY RESULT FROM GPU TO CPU
    hipMemcpy(result.data, d_resultData, WIDTH * HEIGHT * sizeof(uchar4), hipMemcpyDeviceToHost);

    // FREE MEMORY
    hipFree(d_planesData);
    hipFree(d_resultData);

    return result;
}

__global__ void combinePlanesKernel(uchar4* resultData, const uchar4* planesData, int width, int height, int nPlanes) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        auto idx = y * width + x;
        float4 result = make_float4(resultData[idx].x, resultData[idx].y, resultData[idx].z, resultData[idx].w);

        for (int z = 0; z < nPlanes; z++) {
            uchar4 plane = planesData[z * width * height + idx];
            result.x = result.x * (1.0f - ALPHA) + static_cast<float>(plane.x) * ALPHA;
            result.y = result.y * (1.0f - ALPHA) + static_cast<float>(plane.y) * ALPHA;
            result.z = result.z * (1.0f - ALPHA) + static_cast<float>(plane.z) * ALPHA;
            result.w = result.w * (1.0f - ALPHA) + static_cast<float>(plane.w) * ALPHA;
        }

        resultData[idx] = make_uchar4(static_cast<uchar>(result.x), static_cast<uchar>(result.y), static_cast<uchar>(result.z), static_cast<uchar>(result.w));
    }
}

