#include "hip/hip_runtime.h"
#include "renderer.cuh"

Circle* generateCircles(std::size_t n) {
    auto* circles = new Circle[n];
    std::srand(777);
#pragma omp parallel for default(none) shared(circles) firstprivate(n) // PARALLEL GENERATION CIRCLES
    for (int i = 0; i < n; i++) {
        cv::Scalar color(std::rand() % 256, std::rand() % 256, std::rand() % 256, 255);
        cv::Point center(std::rand() % HEIGHT + 1, std::rand() % WIDTH + 1);
        int r = std::rand() % (MAX_RADIUS - MIN_RADIUS) + MIN_RADIUS + 1;
        circles[i] = {color, center, r};
    }
    return circles;
}

double rendererSequential(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            auto circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesSequential(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END

    printf("Sequential time %f sec.\n", time);

    delete[] planes;

    cv::imwrite("../img/seq_" + std::to_string(nPlanes) + ".png", result);
    // cv::waitKey(0);
    return time;
}

cv::Mat combinePlanesSequential(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
    int cn = result.channels();
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double rendererParallel(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesParallel(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END
    printf("Parallel time %f sec.\n", time);

    delete[] planes;

    cv::imwrite("../img/par_" + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat combinePlanesParallel(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
    int cn = result.channels();
#pragma omp parallel for default(none) shared(result, planes) firstprivate(nPlanes, cn) collapse(2)
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            for (int z = 0; z < nPlanes; z++) {
                cv::Mat *src2 = &planes[z];
                for (int c = 0; c < cn; c++)
                    result.data[i * result.step + cn * j + c] =
                            result.data[i * result.step + j * cn + c] * (1 - ALPHA) +
                            src2->data[i * src2->step + j * cn + c] * (ALPHA);
            }
        }
    }
    return result;
}

double rendererCuda(Circle circles[], std::size_t nPlanes, std::size_t nCircles) {
    auto* planes = new cv::Mat[nPlanes];

    // START
    double start = omp_get_wtime();

#pragma omp parallel for default(none) shared(planes, circles) firstprivate(nPlanes, nCircles)
    for (int i = 0; i < nPlanes; i++) {
        planes[i] = cv::Mat(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
        for (int j = 0; j < nCircles; j++) {
            Circle circle = circles[i * nCircles + j];
            cv::circle(planes[i], circle.center, circle.r, circle.color, cv::FILLED, cv::LINE_AA);
        }
    }

    cv::Mat result = combinePlanesCuda(planes, nPlanes);

    double time = omp_get_wtime() - start;
    // END
    printf("Cuda time %f sec.\n", time);

    delete[] planes;

    cv::imshow("TEST", result);
    cv::waitKey(0);

    cv::imwrite("../img/cuda_" + std::to_string(nPlanes) + ".png", result);
    return time;
}

cv::Mat combinePlanesCuda(cv::Mat planes[], std::size_t nPlanes) {
    cv::Mat result(HEIGHT, WIDTH, CV_8UC4, TRANSPARENT);
    int cn = result.channels();

    auto** d_planesData = new uchar*[nPlanes];
    uchar* d_resultData;

    // Initialize pointers on GPU
    hipMalloc((void**)&d_resultData, WIDTH * HEIGHT * cn * sizeof(uchar));
    for (std::size_t i = 0; i < nPlanes; i++) {
        hipMalloc((void**)&d_planesData[i], WIDTH * HEIGHT * cn * sizeof(uchar));
        hipMemcpy(d_planesData[i], planes[i].data, WIDTH * HEIGHT * cn * sizeof(uchar), hipMemcpyHostToDevice);
    }

    // GRID AND BLOCK DIMENSIONS
    dim3 block(16, 16);
    dim3 grid((result.cols + block.x - 1) / block.x, (result.rows + block.y - 1) / block.y);

    // CUDA KERNEL
    combinePlanesKernel<<<grid, block>>>(d_resultData, d_planesData, result.cols, result.rows, (int)nPlanes, cn);
    hipDeviceSynchronize();

    hipMemcpy(result.data, d_resultData, WIDTH * HEIGHT * cn * sizeof(uchar), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < nPlanes; i++)
        hipFree(d_planesData[i]);
    delete[] d_planesData;
    hipFree(d_resultData);

    return result;
}

__global__ void combinePlanesKernel(uchar* resultData, uchar** planesData, int width, int height, int nPlanes, int cn){
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height){
        for (int c = 0; c < cn; c++){
            auto idx = (y * width + x) * cn + c;
            float combinedValue = 0.0f;
            for (int z = 0; z < nPlanes; z++){
                uchar* srcData = planesData[z];
                combinedValue += static_cast<float>(srcData[idx]);
            }
            resultData[idx] = static_cast<uchar>(combinedValue / nPlanes);
        }
    }
}

